#include "hip/hip_runtime.h"
#include "md5.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <cstring>
#include <iostream>

// CUDA 错误检查宏
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// --- 补充宏定义（与md5.h一致） ---
#define s11 7
#define s12 12
#define s13 17
#define s14 22
#define s21 5
#define s22 9
#define s23 14
#define s24 20
#define s31 4
#define s32 11
#define s33 16
#define s34 23
#define s41 6
#define s42 10
#define s43 15
#define s44 21

#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))
#define ROTATELEFT(num, n) (((num) << (n)) | ((num) >> (32-(n))))
#define FF(a, b, c, d, x, s, ac) { \
  (a) += F ((b), (c), (d)) + (x) + ac; \
  (a) = ROTATELEFT ((a), (s)); \
  (a) += (b); \
}
#define GG(a, b, c, d, x, s, ac) { \
  (a) += G ((b), (c), (d)) + (x) + ac; \
  (a) = ROTATELEFT ((a), (s)); \
  (a) += (b); \
}
#define HH(a, b, c, d, x, s, ac) { \
  (a) += H ((b), (c), (d)) + (x) + ac; \
  (a) = ROTATELEFT ((a), (s)); \
  (a) += (b); \
}
#define II(a, b, c, d, x, s, ac) { \
  (a) += I ((b), (c), (d)) + (x) + ac; \
  (a) = ROTATELEFT ((a), (s)); \
  (a) += (b); \
}

__device__ void md5_transform(const unsigned char* msg, int msg_len, unsigned int* state) {
    // 这里只实现单block（<56字节）消息的MD5，适用于短口令
    unsigned int a = 0x67452301;
    unsigned int b = 0xefcdab89;
    unsigned int c = 0x98badcfe;
    unsigned int d = 0x10325476;
    unsigned char block[64] = {0};
    int i;
    for (i = 0; i < msg_len; ++i) block[i] = msg[i];
    block[msg_len] = 0x80;
    unsigned long long bit_len = msg_len * 8;
    memcpy(block + 56, &bit_len, 8);

    unsigned int x[16];
    for (i = 0; i < 16; ++i) {
        x[i] = ((unsigned int)block[i*4]) | ((unsigned int)block[i*4+1] << 8) |
               ((unsigned int)block[i*4+2] << 16) | ((unsigned int)block[i*4+3] << 24);
    }

    // MD5轮函数
    /* Round 1 */
    FF(a, b, c, d, x[0], s11, 0xd76aa478);
    FF(d, a, b, c, x[1], s12, 0xe8c7b756);
    FF(c, d, a, b, x[2], s13, 0x242070db);
    FF(b, c, d, a, x[3], s14, 0xc1bdceee);
    FF(a, b, c, d, x[4], s11, 0xf57c0faf);
    FF(d, a, b, c, x[5], s12, 0x4787c62a);
    FF(c, d, a, b, x[6], s13, 0xa8304613);
    FF(b, c, d, a, x[7], s14, 0xfd469501);
    FF(a, b, c, d, x[8], s11, 0x698098d8);
    FF(d, a, b, c, x[9], s12, 0x8b44f7af);
    FF(c, d, a, b, x[10], s13, 0xffff5bb1);
    FF(b, c, d, a, x[11], s14, 0x895cd7be);
    FF(a, b, c, d, x[12], s11, 0x6b901122);
    FF(d, a, b, c, x[13], s12, 0xfd987193);
    FF(c, d, a, b, x[14], s13, 0xa679438e);
    FF(b, c, d, a, x[15], s14, 0x49b40821);

    /* Round 2 */
    GG(a, b, c, d, x[1], s21, 0xf61e2562);
    GG(d, a, b, c, x[6], s22, 0xc040b340);
    GG(c, d, a, b, x[11], s23, 0x265e5a51);
    GG(b, c, d, a, x[0], s24, 0xe9b6c7aa);
    GG(a, b, c, d, x[5], s21, 0xd62f105d);
    GG(d, a, b, c, x[10], s22, 0x2441453);
    GG(c, d, a, b, x[15], s23, 0xd8a1e681);
    GG(b, c, d, a, x[4], s24, 0xe7d3fbc8);
    GG(a, b, c, d, x[9], s21, 0x21e1cde6);
    GG(d, a, b, c, x[14], s22, 0xc33707d6);
    GG(c, d, a, b, x[3], s23, 0xf4d50d87);
    GG(b, c, d, a, x[8], s24, 0x455a14ed);
    GG(a, b, c, d, x[13], s21, 0xa9e3e905);
    GG(d, a, b, c, x[2], s22, 0xfcefa3f8);
    GG(c, d, a, b, x[7], s23, 0x676f02d9);
    GG(b, c, d, a, x[12], s24, 0x8d2a4c8a);

    /* Round 3 */
    HH(a, b, c, d, x[5], s31, 0xfffa3942);
    HH(d, a, b, c, x[8], s32, 0x8771f681);
    HH(c, d, a, b, x[11], s33, 0x6d9d6122);
    HH(b, c, d, a, x[14], s34, 0xfde5380c);
    HH(a, b, c, d, x[1], s31, 0xa4beea44);
    HH(d, a, b, c, x[4], s32, 0x4bdecfa9);
    HH(c, d, a, b, x[7], s33, 0xf6bb4b60);
    HH(b, c, d, a, x[10], s34, 0xbebfbc70);
    HH(a, b, c, d, x[13], s31, 0x289b7ec6);
    HH(d, a, b, c, x[0], s32, 0xeaa127fa);
    HH(c, d, a, b, x[3], s33, 0xd4ef3085);
    HH(b, c, d, a, x[6], s34, 0x4881d05);
    HH(a, b, c, d, x[9], s31, 0xd9d4d039);
    HH(d, a, b, c, x[12], s32, 0xe6db99e5);
    HH(c, d, a, b, x[15], s33, 0x1fa27cf8);
    HH(b, c, d, a, x[2], s34, 0xc4ac5665);

    /* Round 4 */
    II(a, b, c, d, x[0], s41, 0xf4292244);
    II(d, a, b, c, x[7], s42, 0x432aff97);
    II(c, d, a, b, x[14], s43, 0xab9423a7);
    II(b, c, d, a, x[5], s44, 0xfc93a039);
    II(a, b, c, d, x[12], s41, 0x655b59c3);
    II(d, a, b, c, x[3], s42, 0x8f0ccc92);
    II(c, d, a, b, x[10], s43, 0xffeff47d);
    II(b, c, d, a, x[1], s44, 0x85845dd1);
    II(a, b, c, d, x[8], s41, 0x6fa87e4f);
    II(d, a, b, c, x[15], s42, 0xfe2ce6e0);
    II(c, d, a, b, x[6], s43, 0xa3014314);
    II(b, c, d, a, x[13], s44, 0x4e0811a1);
    II(a, b, c, d, x[4], s41, 0xf7537e82);
    II(d, a, b, c, x[11], s42, 0xbd3af235);
    II(c, d, a, b, x[2], s43, 0x2ad7d2bb);
    II(b, c, d, a, x[9], s44, 0xeb86d391);

    state[0] = a;
    state[1] = b;
    state[2] = c;
    state[3] = d;
}

__global__ void md5_kernel(const char* data, const int* offsets, const int* lengths, unsigned int* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        md5_transform((const unsigned char*)(data + offsets[idx]), lengths[idx], out + idx * 4);
    }
}

void gpu_MD5Hash_batch(const vector<string>& passwords, bit32* states) {
    int n = passwords.size();
    vector<int> offsets(n), lengths(n);
    int total_len = 0;
    for (int i = 0; i < n; ++i) {
        offsets[i] = total_len;
        lengths[i] = passwords[i].size();
        total_len += passwords[i].size();
    }
    vector<char> flat_data(total_len);
    for (int i = 0; i < n; ++i) {
        memcpy(flat_data.data() + offsets[i], passwords[i].data(), passwords[i].size());
    }

    char* d_data = nullptr;
    int* d_offsets = nullptr;
    int* d_lengths = nullptr;
    unsigned int* d_out = nullptr;
    CUDA_CHECK(hipMalloc(&d_data, total_len));
    CUDA_CHECK(hipMalloc(&d_offsets, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_lengths, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_out, n * 4 * sizeof(unsigned int)));
    CUDA_CHECK(hipMemcpy(d_data, flat_data.data(), total_len, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, offsets.data(), n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lengths, lengths.data(), n * sizeof(int), hipMemcpyHostToDevice));

    int block = 512;
    int grid = (n + block - 1) / block;
    md5_kernel<<<grid, block>>>(d_data, d_offsets, d_lengths, d_out, n);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(states, d_out, n * 4 * sizeof(unsigned int), hipMemcpyDeviceToHost));

    hipFree(d_data);
    hipFree(d_offsets);
    hipFree(d_lengths);
    hipFree(d_out);
}
